#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "rec.h"

using namespace std;

typedef struct Band {
  int x1;
  int x2;
  int x3;
  int user;
} Band;

struct BandCmp {
  __host__ __device__
  bool operator() (const Band & o1, const Band & o2) {
    if (o1.x1 < o2.x1) {
      return true;
    } else if (o1.x1 > o2.x1) {
      return false;
    } else {
      if (o1.x2 < o2.x2) {
        return true;
      } else if (o1.x2 > o2.x2) {
        return false;
      } else {
        return o1.x3 < o2.x3;
      }
    }
  }
};

#define IDX(u, i, width) ((u) * (ITEM_COUNT) + i)

#define BLOCK_SIZE 128
#define UPDIV(n,div) ((n + div - 1)/div)

thrust::host_vector<int> uf(USER_SIZE);

// function declarations
static void ann(vector< vector<int> > &data);

static int uf_find(int user) {
  int p = uf[user];
  while (p != uf[p]) {
    p = uf[p];
  }
  uf[user] = p;
  return p;
}

static void uf_union(int self, int other) {
  uf[self] = other;
}

__global__ void mean_kernel(int *compact_data, int *compact_index, double *mean) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // user starts with 1, so tid=0 work on user 1
  if (tid >= USER_SIZE)
    return ;
  int startIdx = compact_index[tid];
  int endIdx;
  if (tid == USER_SIZE-1) {
    endIdx = DATA_SIZE;
  } else {
    endIdx = compact_index[tid+1];
  }
  double sum = 0.0;
  for (int i = startIdx; i < endIdx; i+=2) {
    sum += compact_data[i+1];
  }
  mean[tid] = 2 * sum / (endIdx - startIdx);
}


__global__ void hash_kernel(int *hashed_matrix, int *as, int *bs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int (tid >= ITEM_SIZE)
    return ;
  __shared__ int ass[100];
  __shared__ int bss[100];
  if (tid < 100) {
    ass[tid] = as[tid];
    bss[tid] = bs[tid];
  }
  __syncthreads();

  for (int i = 0; i < 100; i++) {
    hashed_matrix[tid * 100 + i] = (as[i] * tid + bs[i]) % ITEM_SIZE;
  }

}

__global__ void binarize(int *compact_data, int *compact_index, double *mean) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= USER_SIZE)
    return ;
  int startIdx = compact_index[tid];
  int endIdx;
  if (tid == USER_SIZE-1) {
    endIdx = DATA_SIZE;
  } else {
    endIdx = compact_index[tid+1];
  }

  for (int i = startIdx; i < endIdx; i+=2) {
    if (compact_data[i+1] >= mean[tid]) {
      compact_data[i+1] = 1;
    } else {
      compact_data[i+1] = 0;
    }
  }
}

__global__ void lsh_kernel(int *compact_data, int *compact_index,
                           int *hashed_matrix, int *sigs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= USER_SIZE)
    return ;
  // put hashed matrix into shared memory
  __shared__ int hash[ITEM_SIZE * 100];
  if (tid < ITEM_SIZE * 100) {
    hash[tid] = hashed_matrix[tid];
  }
  __syncthreads();

  int u_start = compact_index[tid];
  int u_end;
  if (tid == USER_SIZE - 1) {
    u_end = DATA_SIZE - 1;
  } else {
    u_end = compact_index[tid+1];
  }

  for (int i = u_start; i < u_end; i+=2) {
    int item = compact_data[i];
    int rating = compact_data[i+1];
    if (rating == 0)
      continue;
    for (int j = 0; j < 100; j++) {
      // for all possible hash functions
      int hashed_item = hash[item * 100 + j];
      sigs[tid * 100 + j] = min(sigs[tid * 100 + j], hashed_item);
    }
  }
}


__global__ void recommendation_kernel(int user, int *compact_data, int *compact_index,
                                      double *sim, double *like) {
  user = user - 1;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= USER_SIZE)
    return ;
  int u_start = compact_index[user];
  int v_start = compact_index[tid];
  int u_end, v_end;
  if (user == USER_SIZE - 1) {
    u_end = DATA_SIZE - 1;
  } else {
    u_end = compact_index[user+1];
  }
  if (tid == DATA_SIZE - 1) {
    v_end = DATA_SIZE - 1;
  } else {
    v_end = compact_index[tid+1];
  }
  int i = u_start;
  int j = v_start;
  int item_i, item_j;
  while (i < u_end && j < v_end) {
    item_i = compact_data[i];
    item_j = compact_data[j];
    if (item_i == item_j) {
      i += 2;
      j += 2;
    } else if (item_i < item_j) {
      // possible item_j appear in u's ratings
      i += 2;
    } else {
      // item_i > item_j, item_j won't be rated by u
      like[item_j] += sim[tid] * compact_data[j+1];
      j += 2;
    }
  }
  if (j != v_end) {
    while (j < v_end) {
      item_j = compact_data[j];
      like[item_j] += sim[tid] * compact_data[j+1];
      j += 2;
    }
  }
}

__global__ void similarity_kernel(int user, int *compact_data,
                                  int *compact_index,
                                  double *sim, double *mean) {
  user = user - 1;
  // tid is other_user
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= USER_SIZE)
    return ;
  double a = 0.0;
  double b = 0.0;
  double c = 0.0;
  int commons = 0;
  double u_mean = mean[user];
  double v_mean = mean[tid];
  int u_start = compact_index[user];
  int v_start = compact_index[tid];
  int u_end, v_end;
  if (user == USER_SIZE - 1) {
    u_end = DATA_SIZE - 1;
  } else {
    u_end = compact_index[user+1];
  }
  if (tid == DATA_SIZE - 1) {
    v_end = DATA_SIZE - 1;
  } else {
    v_end = compact_index[tid+1];
  }

  int i = u_start;
  int j = v_start;
  double rui, rvi;
  int item_i, item_j;
  while (i < u_end && j < v_end) {
    item_i = compact_data[i];
    item_j = compact_data[j];
    if (item_i == item_j) { // common item
      commons++;
      rui = compact_data[i+1];
      rvi = compact_data[j+1];
      a += (rui - u_mean) * (rvi - v_mean);
      b += (rui - u_mean) * (rui - u_mean);
      c += (rvi - v_mean) * (rvi - v_mean);
      i += 2;
      j += 2;
    } else if (item_i < item_j) {
      i += 2;
    } else {
      j += 2;
    }
  }
  double answer;
  if (b * c == 0) {
    answer = a;
  } else {
    answer = a / (sqrt(b) * sqrt(c));
  }
  if (commons < 5) {
    answer *= 0.2 * commons;
  }
  sim[tid] = answer;
}

/*
  populate the sim vector
 */
void CUDA_populate_user_sim_vec(int target_user, int *compact_data,
                                int *compact_index, double *sim, int topn, vector< vector<int> > &data) {
  ann(data);
  // calculate the mean for each user
  int *compact_data_cuda;
  int *compact_index_cuda;
  double *mean_cuda;
  double *sim_cuda;
  //double mean[USER_SIZE];
  hipMalloc((void **)&compact_data_cuda, DATA_SIZE*sizeof(int));
  hipMalloc((void **)&compact_index_cuda, USER_SIZE*sizeof(int));
  hipMalloc((void **)&mean_cuda, USER_SIZE*sizeof(double));
  hipMalloc((void **)&sim_cuda, USER_SIZE*sizeof(double));
  hipMemcpy(compact_data_cuda, compact_data, DATA_SIZE*sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(compact_index_cuda, compact_index, USER_SIZE*sizeof(int),
             hipMemcpyHostToDevice);
  int tpb = 1024;
  mean_kernel<<<UPDIV(USER_SIZE, tpb), tpb>>>(compact_data_cuda,
                                               compact_index_cuda,
                                               mean_cuda);
  /* hipMemcpy(mean, mean_cuda, USER_SIZE*sizeof(double), hipMemcpyDeviceToHost); */
  /* for (int i=0; i < USER_SIZE; i++) { */
  /*   cout << "mean for user " << i << ": " << mean[i] << endl; */
  /* } */
  similarity_kernel<<<UPDIV(USER_SIZE, tpb), tpb>>>(target_user,
                                                     compact_data_cuda,
                                                     compact_index_cuda,
                                                     sim_cuda, mean_cuda);
  hipMemcpy(sim, sim_cuda, USER_SIZE*sizeof(double), hipMemcpyDeviceToHost);
  cout << "finished sim" << endl;
  for (int i=0; i < USER_SIZE; i++) {
    cout << "sim " << i << ": " << sim[i] << endl;
  }
  hipFree(sim_cuda);
  hipFree(mean_cuda);
}

static void ann_compact(int *compact_data, int *compact_index) {
  // 1. calculate mean and assign 1/0
  int *compact_data_cuda;
  int *compact_index_cuda;
  double *mean_cuda;
  hipMalloc((void **)&compact_data_cuda, DATA_SIZE*sizeof(int));
  hipMalloc((void **)&compact_index_cuda, USER_SIZE*sizeof(int));
  hipMalloc((void **)&mean_cuda, USER_SIZE*sizeof(double));
  hipMemcpy(compact_data_cuda, compact_data, DATA_SIZE*sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(compact_index_cuda, compact_index, USER_SIZE*sizeof(int),
             hipMemcpyHostToDevice);
  int tpb = 1024;
  mean_kernel<<<UPDIV(USER_SIZE, tpb), tpb>>>(compact_data_cuda,
                                               compact_index_cuda,
                                               mean_cuda);
  binarize<<<UPDIV(USER_SIZE, tpb), tpb>>>(compact_data_cuda,
                                           compact_index_cuda,
                                           mean_cuda);
  // 2. generate the signature matrix
  int n = 100;
  thrust::host_vector<int> sig(USER_SIZE * n);
  thrust::host_vector<int> as(n);
  thrust::host_vector<int> bs(n);

  for (int i = 0; i < n; i++) {
    as[i] = rand() % ITEM_SIZE;
    bs[i] = rand() % ITEM_SIZE;
  }

  thrust::fill(sig.begin(), sig.end(), INT_MAX);

  // map each item to a thread

  int *ass = thrust::raw_pointer_cast(&as[0]);
  int *bss = thrust::raw_pointer_cast(&bs[0]);
  int *sigs= thrust::raw_pointer_cast(&sig[0]);

  int tpb = 1024;

  // hashed items matrix
  // hashed_matrix[i*n + j] : i's item index using hash fxn j
  int hashed_matrix[ITEM_SIZE * n];
  hash_kernel<<<UPDIV(ITEM_SIZE, tpb), tpb>>>(hashed_matrix, ass, bss);

  lsh_kernel<<<UPDIV(ITEM_SIZE, tpb), tpb>>>(compact_data_cuda,
                                             compact_index_cuda,
                                             hashed_matrix, sigs);


  // 3. union find

  thrust::sequence(uf.begin(), uf.end());

  int band_len = 3;

  for (int band = 0; band < n; band+=band_len) {
    thrust::host_vector<Band> v(USER_SIZE);
    for (int i = 0; i < USER_SIZE; i++) {
      Band b;
      b.x1 = sigs[i * 100 + 0];
      b.x2 = sigs[i * 100 + 1];
      b.x3 = sigs[i * 100 + 2];
      b.user = i;
      v.push_back(b);
    }
    thrust::sort(v.begin(), v.end(), BandCmp);
    for (int i = 0; i < UESR_SIZE-1; i++) {
      Band b1 = v[i];
      Band b2 = v[i+1];
      if (b1.x1 == b2.x1 && b1.x2 == b2.x2 && b1.x3 == b2.x3) {
        uf_union(uf_find(b1.user, b2.user), uf_find(b1.user, b2.user));
      }
    }
  }
}



void CUDA_populate_item_like_vec(int user, int *compact_data,
                                 int *compact_index, double *sim,
                                 double *like, int topn) {
  // calculate the mean for each user
  int *compact_data_cuda;
  int *compact_index_cuda;
  double *mean_cuda;
  double *sim_cuda;
  double *like_cuda;
  //double mean[USER_SIZE];
  hipMalloc((void **)&compact_data_cuda, DATA_SIZE*sizeof(int));
  hipMalloc((void **)&compact_index_cuda, USER_SIZE*sizeof(int));
  hipMalloc((void **)&mean_cuda, USER_SIZE*sizeof(double));
  hipMalloc((void **)&sim_cuda, USER_SIZE*sizeof(double));
  hipMalloc((void **)&like_cuda, ITEM_SIZE*sizeof(double));
  hipMemcpy(compact_data_cuda, compact_data, DATA_SIZE*sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(compact_index_cuda, compact_index, USER_SIZE*sizeof(int),
             hipMemcpyHostToDevice);
  int tpb = 1024;
  mean_kernel<<<UPDIV(USER_SIZE, tpb), tpb>>>(compact_data_cuda,
                                               compact_index_cuda,
                                               mean_cuda);
  similarity_kernel<<<UPDIV(USER_SIZE, tpb), tpb>>>(user,
                                                    compact_data_cuda,
                                                    compact_index_cuda,
                                                    sim_cuda, mean_cuda);
  recommendation_kernel<<<UPDIV(USER_SIZE, tpb), tpb>>>(user,
                                                        compact_data_cuda,
                                                        compact_index_cuda,
                                                        sim_cuda, like_cuda);
  hipFree(sim_cuda);
  hipFree(mean_cuda);
}


/**
 * Preprocess the data
 **/
static void ann(vector< vector<int> > &data) {
  // generate the characteristic matrix
  for (int user = 1; user <= USER_SIZE; user++) {
    int total = 0;
    int count = 0;
    for (int i = 1; i <= ITEM_SIZE; i++) {
      if (data[user][i] != 0) {
        total += data[user][i];
        count++;
      }
    }
    int mean = total / count;
    for (int i = 1; i <= ITEM_SIZE; i++) {
      if (data[user][i] > mean) {
        data[user][i] == 1;
      } else {
        data[user][i] == 0;
      }
    }
  }
  // generate the signature matrix using the hash function ax + b mod ITEM_SIZE
  int n = 100;
  int sig[USER_SIZE][n];
  int as[n];
  int bs[n];

  for (int i = 0; i < n; i++) {
    as[i] = rand() % ITEM_SIZE;
    bs[i] = rand() % ITEM_SIZE;
  }

  for (int i = 0; i < USER_SIZE; i++) {
    for (int j = 0; i < n; j++) {
      sig[i][j] = INT_MAX;
    }
  }

  for (int item = 0; item < ITEM_SIZE; item++) {
    // 1. compute permuted rows
    for (int i = 0; i < n; i++) {
      cs[i] = (as[i] * item + bs[i]) % ITEM_SIZE;
    }
    // 2. for each column c:
    for (int user = 0; user < USER_SIZE; user++) {
      if (data[user][item] == 0) {
        // do nothing
      } else {
        for (int x = 1; x < n; x++) {
          sig[user][x] = min(sig[user][x], cs[x]);
        }
      }
    }
  }


  for (int r = 0; i < ITEM_SIZE; i++) {
    for (int hash = 0; hash < n; hash++) {
      for (int user = 1; user < USER_SIZE; user++) {
        sig[hash][user] = cs[hash];
      }
    }
  }

  // initialize the union find data structure

  for (int i = 0; i < USER_SIZE; i++) {
    uf[i] = i;
  }

  int band_len = 3;

  for (int band = 0; band < n; band+=band_len) {
    for (int self = 0; self < USER_SIZE; self++) {
      for (int other = self+1; other < USER_SIZE; other++) {
        int a[3];
        int b[3];
        a[0] = sig[band*band_len][self];
        b[0] = sig[band*band_len][other];
        a[1] = sig[band*band_len+1][self];
        b[1] = sig[band*band_len+1][other];
        a[2] = sig[band*band_len+2][self];
        b[2] = sig[band*band_len+2][other];
        if (a[0]==b[0] && a[1]==b[1] && a[2]==b[2]) {
          // find(self)
          int u = uf[self];
          uf_union(uf_find(self), uf_find(other));
        }
      }
    }
  }
}

/*
  ANN Interface
 */
thrust::host_vector<int> get_neighbors(int user) {
  thrust::host_vector<int> neighbors;
  for (int other = 0; other < USER_SIZE; other++) {
    if (uf_find(user) == uf_find(other)) {
      neighbors.push_back(other);
    }
  }
  return neighbors;
}
