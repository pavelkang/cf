#include <stdio.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#define USER_COUNT 944
#define ITEM_COUNT 2000
#define IDX(u, i, width) ((u) * (ITEM_COUNT) + i)

#define BLOCK_SIZE 128
#define UPDIV(n,div) ((n + div - 1)/div)

__global__ void recommendation_kernel(double *sim, double *users, double *out, const int user, const int N, const int M) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ double sim_shared[BLOCK_SIZE];

    double sum = 0;
    for (int j = 0; j < UPDIV(N, BLOCK_SIZE); j++) {

        sim_shared[threadIdx.x] = ((j * BLOCK_SIZE + threadIdx.x) < N)
                                  ? sim[(threadIdx.x + j * BLOCK_SIZE)]
                                  : 0.0;
        __syncthreads();


        if (tid < M && users[IDX(user, tid, M)] == 0) {
            #pragma unroll
            for (int e = 0; e < BLOCK_SIZE; e++) {
                int i = e + BLOCK_SIZE * j;
                if (i < N) {
                    sum += sim_shared[e] * users[IDX(i, tid, M)];
                }
            }
        }
        __syncthreads();
    }
    /*
    if (users[IDX(user, tid, M)] == 0)  {
        for (int i = 0; i < USER_COUNT; i++) {
            sum += sim[i] * users[IDX(i, tid, M)];
        }
    }
    */


    if (tid < M) out[tid] = sum;

}


__global__ void similarity_kernel(double * mean, double *users, double *out, const int user,const int N, const int M) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //printf("tid %d\n", tid);
    double a = 0.0;
    double b = 0.0;
    double c = 0.0;
    int commons = 0;

    double v_mean = mean[user];
    __shared__ double user_rate_shared[BLOCK_SIZE];

    for (int j = 0; j < UPDIV(N, BLOCK_SIZE); j++) {

        if ((j * BLOCK_SIZE + threadIdx.x) < N) {
            user_rate_shared[threadIdx.x] = users[IDX(user, (threadIdx.x + j * BLOCK_SIZE), N)];
        } else {
            user_rate_shared[threadIdx.x] = 0.f;
        }
        /*
        user_rate_shared[threadIdx.x] = ((j * BLOCK_SIZE + threadIdx.x) < N)
                                        ? users[IDX(user, (threadIdx.x + j * BLOCK_SIZE) ,N)]
                                        : 0.0;
                                        */
        __syncthreads();

        #pragma unroll

        for (int e = 0; e < BLOCK_SIZE; e++) {
            int i = e + BLOCK_SIZE * j;

            if ((tid < M) && (user_rate_shared[e] != 0) && (users[IDX(tid, (i), N)] != 0)) {
                double u_mean = mean[tid];
                commons++;
                double rui = users[IDX(tid,i,N)];
                double rvi = user_rate_shared[e];
                a += (rui - u_mean)*(rvi - v_mean);
                b += (rui-u_mean)*(rui-u_mean);
                c += (rvi-v_mean)*(rvi-v_mean);

            }
        }
        __syncthreads();
    }

    double answer;
    if (b*c == 0) {
        answer = a;
    } else {
        answer = a / (sqrt(b) * sqrt(c));
    }
    // fix pearson
    if (commons < 5) {
        answer *= 0.2 * commons;
    }

    if (tid < M) out[tid] = answer;
}

__global__ void mean_kernel(double *users, double *out, const int N, const int M) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    double sum = 0;
    int count = 0;
    if (tid > M) return;
    for (int i = 0; i < N; i++) {
        double rate = users[tid * N + i];
        sum += rate;
        if (rate) count++;
    }
    out[tid] = sum / count;
}

void CUDA_get_similar_users(double *um, int user, double *similarity_copy, int topn = 5) {

    double *d_users;
    double *d_mean;
    double *d_out;
    hipMalloc((void **)&d_users, USER_COUNT * ITEM_COUNT * sizeof(double));
    hipMalloc((void **)&d_mean, USER_COUNT * sizeof(double));
    hipMalloc((void **)&d_out , USER_COUNT * sizeof(double));

    hipMemcpy(d_users, um, USER_COUNT * ITEM_COUNT * sizeof(double), hipMemcpyHostToDevice);
    mean_kernel<<<USER_COUNT / 1024 + 1, 1024 >>>(d_users, d_mean, ITEM_COUNT, USER_COUNT);

    dim3 dim_grid(UPDIV(USER_COUNT, BLOCK_SIZE));
    dim3 dim_block(BLOCK_SIZE);

    similarity_kernel<<<dim_grid, dim_block>>>(d_mean, d_users, d_out, user, ITEM_COUNT, USER_COUNT);
    hipMemcpy(similarity_copy, d_out, USER_COUNT * sizeof(double), hipMemcpyDeviceToHost);

    similarity_copy[user] = 0;

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    hipFree(d_users);
    hipFree(d_mean);
    hipFree(d_out);
}

void CUDA_get_recommendations(double *um, int user, double *recommendation_copy, int topn = 5) {

    double *d_users;
    double *d_mean;
    double *d_sim;
    double *d_out;
    hipMalloc((void **)&d_users, USER_COUNT * ITEM_COUNT * sizeof(double));
    hipMalloc((void **)&d_mean, USER_COUNT * sizeof(double));
    hipMalloc((void **)&d_sim , USER_COUNT * sizeof(double));
    hipMalloc((void **)&d_out , ITEM_COUNT * sizeof(double));

    hipMemcpy(d_users, um, USER_COUNT * ITEM_COUNT * sizeof(double), hipMemcpyHostToDevice);
    mean_kernel<<<USER_COUNT / 1024 + 1, 1024 >>>(d_users, d_mean, ITEM_COUNT, USER_COUNT);

    dim3 dim_grid(UPDIV(USER_COUNT, BLOCK_SIZE));
    dim3 dim_block(BLOCK_SIZE);

    similarity_kernel<<<dim_grid, dim_block>>>(d_mean, d_users, d_sim, user, ITEM_COUNT, USER_COUNT);

    dim3 rdim_grid(UPDIV(ITEM_COUNT, BLOCK_SIZE));
    recommendation_kernel<<<rdim_grid, dim_block>>>(d_sim, d_users, d_out, user, USER_COUNT, ITEM_COUNT);

    hipMemcpy(recommendation_copy, d_out, ITEM_COUNT * sizeof(double), hipMemcpyDeviceToHost);

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    hipFree(d_users);
    hipFree(d_mean);
    hipFree(d_sim);
    hipFree(d_out);
}
